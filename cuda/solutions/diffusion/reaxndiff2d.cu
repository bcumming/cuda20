#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#include <hip/hip_runtime.h>

#include "util.hpp"
#include "hip_stream.hpp"

// 2D diffusion example
// the grid has a fixed width of nx=128
// the use specifies the height, ny, as a power of two
// note that nx and ny have 2 added to them to account for halos

template <typename T>
void fill_gpu(T *v, T value, int n);

void write_to_file(int nx, int ny, double* data);

template <typename T>
void fill_random_ones(T* v, double probability, int n);

template <typename T>
void fill_hole(T* v, int n, int nx, int ny);

__global__
void reactiondiffusion_u(double *u0, double *u1, double *v0, int nx, int ny, double dt) {
    int i = threadIdx.x + blockDim.x*blockIdx.x + 1;
    int j = threadIdx.y + blockDim.y*blockIdx.y + 1;

    double d_u = 0.01;
    double f   = 0.035;

    if (i<nx-1 && j<ny-1) {
        int pos = nx*j + i;
        double v0_pos = v0[pos];
        double u0_pos = u0[pos];

        u1[pos] = u0_pos + dt * (d_u*(-4.*u0_pos
                     + u0[pos-1] + u0[pos+1]
                     + u0[pos-nx] + u0[pos+nx])
                     - u0_pos*v0_pos*v0_pos + f*(1 - u0_pos));

    }
}

__global__
void reactiondiffusion_v(double *v0, double *v1, double *u0, int nx, int ny, double dt) {
    int i = threadIdx.x + blockDim.x*blockIdx.x + 1;
    int j = threadIdx.y + blockDim.y*blockIdx.y + 1;

    double d_v = 0.005;
    double f   = 0.035;
    double k   = 0.065;

    if (i<nx-1 && j<ny-1) {
        int pos = nx*j + i;
        double v0_pos = v0[pos];
        double u0_pos = u0[pos];

        v1[pos] = v0_pos + dt * (d_v*(-4.*v0_pos
                        + v0[pos-1] + v0[pos+1]
                        + v0[pos-nx] + v0[pos+nx])
                        + u0_pos*v0_pos*v0_pos - v0_pos*(f + k));

    }
}
// TODO : implement stencil using 2d launch configuration
// NOTE : i-major ordering, i.e. x[i,j] is indexed at location [i+j*nx]
//  for(i=1; i<nx-1; ++i) {
//    for(j=1; j<ny-1; ++j) {
//        x1[i,j] = x0[i,j] + dt * (-4.*x0[i,j]
//                   + x0[i,j-1] + x0[i,j+1]
//                   + x0[i-1,j] + x0[i+1,j]);
//    }
//  }

int main(int argc, char** argv) {
    // set up parameters
    // first argument is the y dimension = 2^arg
    size_t pow    = read_arg(argc, argv, 1, 8);
    // second argument is the number of time steps
    size_t nsteps = read_arg(argc, argv, 2, 100);

    // set domain size
    size_t nx = (1 << pow)+2;
    size_t ny = (1 << pow)+2;
    double dt = 0.1;

    std::cout << "\n## " << nx << "x" << ny
              << " for " << nsteps << " time steps"
              << " (" << nx*ny << " grid points)"
              << std::endl;

    // allocate memory on device and host
    // note : allocate enough memory for the halo around the boundary
    auto buffer_size = nx*ny;
    double *u_host = malloc_host<double>(buffer_size);
    double *u0     = malloc_device<double>(buffer_size);
    double *u1     = malloc_device<double>(buffer_size);

    double *v_host = malloc_host<double>(buffer_size);
    double *v0     = malloc_device<double>(buffer_size);
    double *v1     = malloc_device<double>(buffer_size);

    // set random initial conditions of 0s and 1s everywhere
    fill_random_ones(u_host, 0.001, buffer_size);
    fill_hole(u_host, buffer_size, nx, ny);
    copy_to_device(u_host, u0, buffer_size);
    copy_to_device(u_host, u1, buffer_size);

    fill_random_ones(v_host, 0.01, buffer_size);
    copy_to_device(v_host, v0, buffer_size);
    copy_to_device(v_host, v1, buffer_size);

    // set boundary conditions of 1 on south border
    // fill_gpu(x0, 1., nx);
    // fill_gpu(x1, 1., nx);
    // fill_gpu(x0+nx*(ny-1), 1., nx);
    // fill_gpu(x1+nx*(ny-1), 1., nx);

    hip_stream stream;
    hip_stream copy_stream();
    auto start_event = stream.enqueue_event();

    // time stepping loop
    auto find_num_blocks = [](int x, int bdim) {return (x+bdim-1)/bdim;};
    dim3 block_dim(16, 16);
    int nbx = find_num_blocks(nx-2, block_dim.x);
    int nby = find_num_blocks(ny-2, block_dim.y);
    dim3 grid_dim(nbx, nby);
    
    for(auto step=0; step<nsteps; ++step) {
        // TODO: launch the diffusion kernel in 2D
        reactiondiffusion_u<<<grid_dim, block_dim>>>(u0, u1, v0, nx, ny, dt);

        reactiondiffusion_v<<<grid_dim, block_dim>>>(v0, v1, u0, nx, ny, dt);

        std::swap(u0, u1);
        std::swap(v0, v1);
    }
    auto stop_event = stream.enqueue_event();
    stop_event.wait();

    copy_to_host<double>(u0, u_host, buffer_size);
    copy_to_host<double>(v0, v_host, buffer_size);

    double time = stop_event.time_since(start_event);

    std::cout << "## " << time << "s, "
              << nsteps*(nx-2)*(ny-2) / time << " points/second"
              << std::endl << std::endl;

    std::cout << "writing to output.bin/bov" << std::endl;
    write_to_file(nx, ny, u_host);

    return 0;
}

template <typename T>
__global__
void fill(T *v, T value, int n) {
    int tid  = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid<n) {
        v[tid] = value;
    }
}

template <typename T>
void fill_gpu(T *v, T value, int n) {
    auto block_dim = 192ul;
    auto grid_dim = n/block_dim + (n%block_dim ? 1 : 0);

    fill<T><<<grid_dim, block_dim>>>(v, value, n);
}

template <typename T>
void fill_random_ones(T* v, double probability, int n)
{
    probability = fmod(probability, 1); // ensure it is between 0 and 1;

    std::srand(std::time(nullptr));

    for (int i = 0; i < n; i++)
    {
        double rnd = ((double) rand() / (RAND_MAX));
        if (rnd > probability)
        {
            v[i] = 0;
        }
        else
        {
            v[i] = 0.2;
        }
    }
}

template <typename T>
void fill_hole(T* v, int n, int nx, int ny)
{
    for (int i = 0; i < n; i++)
    {
        int x = i%nx;
        int y = (i - x)/ny;

        if (x > 0.4*nx && x < 0.6*nx && y > 0.4*ny && y < 0.6*ny)
        {
            v[i] = 0;
        }
    }
}

void write_to_file(int nx, int ny, double* data) {
    {
        FILE* output = fopen("output.bin", "w");
        fwrite(data, sizeof(double), nx * ny, output);
        fclose(output);
    }

    std::ofstream fid("output.bov");
    fid << "TIME: 0.0" << std::endl;
    fid << "DATA_FILE: output.bin" << std::endl;
    fid << "DATA_SIZE: " << nx << " " << ny << " 1" << std::endl;;
    fid << "DATA_FORMAT: DOUBLE" << std::endl;
    fid << "VARIABLE: phi" << std::endl;
    fid << "DATA_ENDIAN: LITTLE" << std::endl;
    fid << "CENTERING: nodal" << std::endl;
    fid << "BRICK_SIZE: 1.0 1.0 1.0" << std::endl;
}